#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cstdlib>
#include <cmath>
#include <algorithm>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <assert.h>


#define cdpErrchk(ans) { cdpAssert((ans), __FILE__, __LINE__); }
__device__ void cdpAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) assert(0);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define BLOCK_SIZE 16
#define TILE_SIZE 16


#define LAPLACIAN_GAUSSIAN 0
#define GAUSSIAN_KERNEL_SIZE 5
#define GAUSSIAN_SIGMA 1.4
#define SHARED 1
#define TILED 1


#define MAX_THRESHOLD_MULT 0.2//*255
#define MIN_THRESHOLD_MULT 0.01 //*255
#define NON_MAX_SUPPR_THRESHOLD 1

__constant__ float sobel_h_constant[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
__constant__ float sobel_v_constant[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
__constant__ float gaussian_filter_constant[GAUSSIAN_KERNEL_SIZE*GAUSSIAN_KERNEL_SIZE];
__constant__


__global__ void apply_filter_global(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel){
	
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < height && j < width){
		float sum = 0.0f; // Initialize sum to 0 inside the loop

		for (int k = 0; k < kernel_size; k++)
		{
			for (int m = 0; m < kernel_size; m++)
			{
				int input_row = i + (k - 1);
				int input_col = j + (m - 1);

				// Check if the indices are within bounds
				if (input_row >= 0 && input_row < height && input_col >= 0 && input_col < width) {
					sum += kernel[k * kernel_size + m] * input[input_row * width + input_col];
				}	
			}
		}

		output[i * width + j] = abs(sum);
	}
}


__global__ void apply_filter_shared(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel){
	extern __shared__ float kernel_shared[];

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIdx.x < kernel_size && threadIdx.y < kernel_size) {
		kernel_shared[threadIdx.y * kernel_size + threadIdx.x] = kernel[threadIdx.y * kernel_size + threadIdx.x];
	}

	__syncthreads(); // Ensure all threads have finished copying to shared memory

	if (i < height && j < width) {
		float sum = 0;
		for (int k = 0; k < kernel_size; k++) {
			for (int m = 0; m < kernel_size; m++) {
				int input_row = i + (k - 1);
				int input_col = j + (m - 1);

				// Check if the indices are within bounds
				if (input_row >= 0 && input_row < height && input_col >= 0 && input_col < width) {
					sum += kernel_shared[k * kernel_size + m] * input[input_row * width + input_col];
				}
			}
		}
		

		output[i * width + j] = abs(sum);
	}
}


//0 gaussian
//1 sobel_h
//2 sobel_v
__global__ void apply_filter_shared_tiled(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, int code){
	
	__shared__ uint8_t input_shared[TILE_SIZE][TILE_SIZE];

	float* kernel_shared;
	if(code == 0){
		kernel_shared= gaussian_filter_constant;
	} else if(code == 1){
		kernel_shared= sobel_h_constant;
	} else if(code == 2){
		kernel_shared= sobel_v_constant;
	}


	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIdx.x < kernel_size && threadIdx.y < kernel_size) {
		input_shared[threadIdx.y][threadIdx.x] = input[i*width + j];
	}

	if (threadIdx.x == 0 && threadIdx.y == 0){
		if(input_shared[threadIdx.y][threadIdx.x] != 0)
		printf("i = %d, j = %d, input_shared -> %d\n", i, j, input_shared[threadIdx.y][threadIdx.x]);
	}


	__syncthreads(); // Ensure all threads have finished copying to shared memory


	if (i < height && j < width) {
		float sum = 0;

		// printf("%d, ", input[i*width + j]);
		//printf("%f, %f, %f, %f, %f, %f, %f, %f, %f\n", kernel_shared[0], kernel_shared[1], kernel_shared[2], kernel_shared[3], kernel_shared[4], kernel_shared[5], kernel_shared[6], kernel_shared[7], kernel_shared[8]);
		for (int k = 0; k < kernel_size; k++) {
			for (int m = 0; m < kernel_size; m++) {
				int input_row = i + (k - 1);
				int input_col = j + (m - 1);

				// Check if the indices are within bounds
				if (input_row >= 0 && input_row < height && input_col >= 0 && input_col < width) {
					sum += kernel_shared[k * kernel_size + m] * input_shared[threadIdx.y + (k - 1)][threadIdx.x + (m - 1)];
					if(threadIdx.x == 1 && threadIdx.y == 1){
						//printf("k = %d, m= %d, kernel_shared -> %f, input -> %d\n",k,m, kernel_shared[k * kernel_size + m], input[input_row * width + input_col]);
					}
				}
			}
		}
		

		output[i * width + j] = abs(sum);
	}


}

__global__ void convert_to_greyscale(int height, int width, uint8_t *img, uint8_t *grey_img)
{

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < height && j < width){
		auto b = img[i*width*3 + j*3 + 0];
		auto g = img[i*width*3 + j*3 + 1];
		auto r = img[i*width*3 + j*3 + 2];

		int average = (int)(0.3*r + 0.59*g + 0.11*b); // Adjust the weights for each channel

		grey_img[i*width + j] = average;
	}
}

__global__ void compute_magnitude_and_gradient(int height, int width, uint8_t *Ix, uint8_t *Iy, uint8_t *mag, float *grad){


	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < height && j < width){
		float dx = Ix[i*width+j];
		float dy = Iy[i*width+j];
		mag[i*width+j] = round(sqrt(dx*dx+dy*dy));
		float angle = atan2(dy, dx)*180/M_PI;
		grad[i*width+j] = angle < 0 ? angle+180 : angle;
	}
	

}

__global__ void non_maximum_suppression_non_interpolated(int height, int width, uint8_t *suppr_mag, uint8_t *mag, float* grad){

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	// here shared memory could be used to save some global loads from mag

	if(i<height && j<width){
		int q = 255;
		int r = 255;

		float grad_ij = grad[i*width+j];

		//angle 0
		if (0 <= grad_ij < 22.5 || 157.5 <= grad_ij <= 180){
			q = mag[i*width + j+1];
			r = mag[i*width + j-1];
		}
		//angle 45
		else if (22.5 <= grad_ij < 67.5){
			q = mag[(i+1)*width + j-1];
			r = mag[(i-1)*width + j+1];
		}
		//angle 90
		else if (67.5 <= grad_ij < 112.5){
			q = mag[(i+1)*width + j];
			r = mag[(i-1)*width + j];
		}
		//angle 135
		else if (112.5 <= grad_ij < 157.5){
			q = mag[(i-1)*width + j-1];
			r = mag[(i+1)*width + j+1];
		}

		float mag_ij = NON_MAX_SUPPR_THRESHOLD*mag[i*width + j];

		if (mag_ij >= q && mag_ij >= r){
			suppr_mag[i*width + j] = mag_ij;
		} else {
			suppr_mag[i*width + j] = 0;
		}
	}


}

__global__ void float_threshold(int height, int width,  uint8_t *pixel_classification,  uint8_t *suppr_mag){

	float high_threshold = MAX_THRESHOLD_MULT*255;
	float low_threshold = MIN_THRESHOLD_MULT*255;

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < height && j < width){
		if(suppr_mag[i*width+j] >= high_threshold){
			// strong pixels
			pixel_classification[i*width+j] = 255;
		} else if (suppr_mag[i*width+j] < low_threshold){
			// non relevant pixels
			pixel_classification[i*width+j] = 0;
		} else {
			// weak pixels
			pixel_classification[i*width+j] = 25;
		}
	}
}

__global__ void hysteresis(int height, int width, uint8_t *pixel_classification){

	// TODO: implement like in paper
	
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;


	// printf("i = %d, j = %d\n", i, j);
	if(i < height && j < width){
		if(pixel_classification[i*width+j] == 25){
			if(pixel_classification[(i+1)*width+j-1] == 255 || pixel_classification[(i+1)*width+j] == 255 || pixel_classification[(i+1)*width+j+1] == 255 ||
			pixel_classification[i*width+j-1] == 255 || pixel_classification[i*width+j+1] == 255 || pixel_classification[(i-1)*width+j-1] == 255 ||
			pixel_classification[(i-1)*width+j] == 255 || pixel_classification[(i-1)*width+j+1] == 255){
				pixel_classification[i*width + j] = 255;
			} else {
				pixel_classification[i*width + j] = 0;
			}
		}
	}

}


__global__ void apply_dilation(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= 1 && i < height - 1 && j >= 1 && j < width - 1) {
        uint8_t max_val = 0;
        for (int k = 0; k < kernel_size; k++) {
            for (int m = 0; m < kernel_size; m++) {
                // Ensure valid indices for input array
                int input_index = (i + (k - 1)) * width + j + (m - 1);
                    auto value = kernel[k * kernel_size + m] * input[input_index];
                    max_val = max_val > value ? max_val : value;
                
            }
        }
        output[i * width + j] = abs(max_val);
    }
}



__global__ void apply_erosion(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel)
{

int i = blockIdx.y * blockDim.y + threadIdx.y;
int j = blockIdx.x * blockDim.x + threadIdx.x;

if(i < height && j < width){
	uint8_t min_val = 255;
	for (int k = 0; k < kernel_size; k++) {
		for (int m = 0; m < kernel_size; m++) {

			min_val = min_val < kernel[k * kernel_size + m] * input[(i + (k - 1)) * width + j + (m - 1)] ? min_val : kernel[k * kernel_size + m] * input[(i + (k - 1)) * width + j + (m - 1)];

		}
	}
	output[i * width + j] = abs(min_val);

}
}


// void measure_time(bool start, FILE* file_times, std::string name){
// 	static std::chrono::system_clock::time_point start_time;
// 	static std::chrono::system_clock::time_point end_time;
// 	if(start){
// 		start_time = std::chrono::system_clock::now();
// 	} else {
// 		end_time = std::chrono::system_clock::now();
// 		std::chrono::duration<float> duration = end_time - start_time;
// 		fprintf(file_times, "%s: %f \n", name.c_str(), duration.count());
// 	}
// }

float* get_gaussian_filter (int kernel_size, float sigma){

	kernel_size = kernel_size%2 == 0 ? kernel_size-1 : kernel_size;

	float* gaussian_filter = (float*)malloc(kernel_size*kernel_size*sizeof(float));
	float sum = 0.0;
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
			gaussian_filter_constant[i*kernel_size + j] = gaussian_filter[i*kernel_size + j];
			sum += gaussian_filter[i*kernel_size + j];
		}
	}
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] /= sum;
			gaussian_filter_constant[i*kernel_size + j] /= sum;
		}
	}
	return gaussian_filter;

}

float* get_gaussian_laplacian_filter (int kernel_size, float sigma){
	float* gaussian_filter = (float*)malloc(kernel_size*kernel_size*sizeof(float));
	float sum = 0.0;
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] = (((i*i+j*j)/(2*sigma*sigma))-1)*exp(-(i*i+j*j)/(2*sigma*sigma))/(M_PI*sigma*sigma*sigma*sigma);
			sum += gaussian_filter[i*kernel_size + j];
		}
	}
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] /= sum;
		}
	}
	return gaussian_filter;


}



int main(int argc, char *argv[])
{
    //Cuda definitions
    const int blocksize = BLOCK_SIZE;
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    int device;
    struct hipDeviceProp_t properties;
    

    hipError_t err = hipSuccess;
    hipDeviceProp_t deviceProp;
    int devID = 0;
    auto error = hipGetDevice(&devID);

    if (error != hipSuccess) {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited) {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_FAILURE);
    }

    if (error != hipSuccess) {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    dim3 threads, grid;

    
    //image definitions
    int width, height, bpp;
    
	auto img_fname = argc>=2 ? argv[1] : "image.png";

	system("mkdir -p output_GPU");
	auto file_times = fopen("./output/times.txt", "w");
	

    //program starts

	uint8_t* rgb_image = stbi_load(img_fname, &width, &height, &bpp, 3);
    uint8_t* rgb_image_d;
    hipMalloc(&rgb_image_d, width*height*3);
    hipMemcpy(rgb_image_d, rgb_image, width*height*3, hipMemcpyHostToDevice);

	threads = dim3(blocksize, blocksize);
    grid = dim3((width + threads.x - 1) / threads.x , (height + threads.y - 1) / threads.y);
    printf("CUDA kernel launch with %d blocks of %d threads\n", grid.x * grid.y, threads.x * threads.y);



    std::cout<<"image: "<<img_fname<<std::endl;
	std::cout<<width<<" "<<height<<std::endl;

    //Stop here

	// Convert to greyscale
	uint8_t* grey_image;
	uint8_t* grey_image_d;
	grey_image = (uint8_t*)malloc(width*height);
	hipMalloc(&grey_image_d, width*height);

	//measure_time(true, file_times, "convert_to_greyscale");
	// convert_to_greyscale(height, width, rgb_image, grey_image);
	convert_to_greyscale<<<grid, threads>>>(height, width, rgb_image_d, grey_image_d);
	hipMemcpy(grey_image, grey_image_d, width*height, hipMemcpyDeviceToHost);

	stbi_image_free(rgb_image);
	hipFree(rgb_image_d);
	stbi_write_png("./output_GPU/0_image_grey.png", width, height, 1, grey_image, width);

	
	// Apply Gaussian filtering
	
	auto kernel_size = GAUSSIAN_KERNEL_SIZE;
	float sigma = GAUSSIAN_SIGMA;
	#if LAPLACIAN_GAUSSIAN
		float* gaussian_filter = get_gaussian_laplacian_filter(kernel_size, sigma);
		#if SHARED && TILED
			hipMemcpyToSymbol(HIP_SYMBOL(gaussian_filter_constant), gaussian_filter, kernel_size*kernel_size*sizeof(float));
		#endif
	#else
		float* gaussian_filter = get_gaussian_filter(kernel_size, sigma);
	#endif
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			std::cout<<gaussian_filter[i*kernel_size + j]<<" ";
		}
		std::cout<<std::endl;
	}

	uint8_t* gaussian_image;
	uint8_t* gaussian_image_d;
	float* gaussian_filter_d;
    gaussian_image = (uint8_t*)malloc(width*height);
	hipMalloc(&gaussian_image_d, width*height);
	hipMalloc(&gaussian_filter_d, kernel_size*kernel_size*sizeof(float));
	hipMemcpy(gaussian_filter_d, gaussian_filter, kernel_size*kernel_size*sizeof(float), hipMemcpyHostToDevice);	

	#if SHARED && TILED
		auto tile_size_alt = TILE_SIZE + kernel_size - 1;
		printf("%d\n", tile_size_alt);
		printf("shared and tiled\n");
		apply_filter_shared_tiled<<<grid, threads, sizeof(uint8_t)*(tile_size_alt * tile_size_alt)>>>(kernel_size, height, width, gaussian_image_d, grey_image_d, 0);
	#else
	#if SHARED	
		printf("shared\n");
		apply_filter_shared<<<grid, threads, kernel_size*kernel_size*sizeof(float)>>>(kernel_size, height, width, gaussian_image_d, grey_image_d, gaussian_filter_d);
	#else
		printf("global\n");
		apply_filter_global<<<grid, threads>>>(kernel_size, height, width, gaussian_image_d, grey_image_d, gaussian_filter_d);
	#endif
	#endif

	hipMemcpy(gaussian_image, gaussian_image_d, width*height, hipMemcpyDeviceToHost);
	stbi_image_free(grey_image);
	stbi_write_png("./output_GPU/0_image_gaussian.png", width, height, 1, gaussian_image, width);

	

	//Apply 3x3 Sobel filtering
	float sobel_h[9] = {-1.0f, 0.0f, 1.0f, -2.0f, 0.0f, 2.0f, -1.0f, 0.0f, 1.0f};
	float sobel_v[9] = {1.0f, 2.0f, 1.0f, 0.0f, 0.0f, 0.0f, -1.0f, -2.0f, -1.0f};
	uint8_t* sobel_image_h;
	uint8_t* sobel_image_v;
	uint8_t* sobel_image_h_d;
	uint8_t* sobel_image_v_d;
	float* sobel_h_d;
	float* sobel_v_d;

	sobel_image_h = (uint8_t*)malloc(width*height);
	sobel_image_v = (uint8_t*)malloc(width*height);
	hipMalloc(&sobel_image_h_d, width*height);
	hipMalloc(&sobel_image_v_d, width*height);
	hipMalloc(&sobel_h_d, 9*sizeof(float));
	hipMalloc(&sobel_v_d, 9*sizeof(float));

	hipMemcpy(sobel_h_d, sobel_h, 9*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(sobel_v_d, sobel_v, 9*sizeof(float), hipMemcpyHostToDevice);


	#if SHARED && TILED
		apply_filter_shared_tiled<<<grid, threads, sizeof(uint8_t)*(TILE_SIZE+2)*(TILE_SIZE+2)>>>(3, height, width, sobel_image_h_d, gaussian_image_d, 1);
		apply_filter_shared_tiled<<<grid, threads, sizeof(uint8_t)*(TILE_SIZE+2)*(TILE_SIZE+2)>>>(3, height, width, sobel_image_v_d, gaussian_image_d, 2);
	#else
	#if SHARED
		apply_filter_shared<<<grid, threads, 9*sizeof(float)>>>(3, height, width, sobel_image_h_d, gaussian_image_d, sobel_h_d);
		apply_filter_shared<<<grid, threads, 9*sizeof(float)>>>(3, height, width, sobel_image_v_d, gaussian_image_d, sobel_v_d);
	#else
		apply_filter_global<<<grid, threads>>>(3, height, width, sobel_image_h_d, gaussian_image_d, sobel_h_d);
		apply_filter_global<<<grid, threads>>>(3, height, width, sobel_image_v_d, gaussian_image_d, sobel_v_d);
	#endif
	#endif

	hipMemcpy(sobel_image_h, sobel_image_h_d, width*height, hipMemcpyDeviceToHost);
	hipMemcpy(sobel_image_v, sobel_image_v_d, width*height, hipMemcpyDeviceToHost);

	stbi_image_free(gaussian_image);
	stbi_write_png("./output_GPU/1_image_sobel_h.png", width, height, 1, sobel_image_h, width);
	stbi_write_png("./output_GPU/1_image_sobel_v.png", width, height, 1, sobel_image_v, width);




	// // Calculate magnitude and gradient direction
    

	float* gradient_direction;
	float* gradient_direction_d;
	uint8_t* magnitude;
	uint8_t* magnitude_d;
	
	gradient_direction = (float*)malloc(width*height*sizeof(float));
	magnitude = (uint8_t*)malloc(width*height);
	hipMalloc(&gradient_direction_d, width*height*sizeof(float));
	hipMalloc(&magnitude_d, width*height);

	compute_magnitude_and_gradient<<<grid, threads>>>(height, width, sobel_image_h_d, sobel_image_v_d, magnitude_d, gradient_direction_d);

	hipMemcpy(gradient_direction, gradient_direction_d, width*height*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(magnitude, magnitude_d, width*height, hipMemcpyDeviceToHost);

	stbi_image_free(sobel_image_v);
	stbi_image_free(sobel_image_h);
	stbi_write_png("./output_GPU/2_gradient_direction.png", width, height, 1, gradient_direction, width);
	stbi_write_png("./output_GPU/2_magnitude.png", width, height, 1, magnitude, width);

	// // Non-maximum suppression
	uint8_t* suppr_mag;
	uint8_t* suppr_mag_d;
	suppr_mag = (uint8_t*)malloc(width*height);
	hipMalloc(&suppr_mag_d, width*height);

	non_maximum_suppression_non_interpolated<<<grid, threads>>>(height, width, suppr_mag_d, magnitude_d, gradient_direction_d);

	hipMemcpy(suppr_mag, suppr_mag_d, width*height, hipMemcpyDeviceToHost);

	stbi_image_free(magnitude);
	stbi_image_free(gradient_direction);
	stbi_write_png("./output_GPU/3_nonmax_suppr.png", width, height, 1, suppr_mag, width);

	// // float thresholding and edge tracking by hysteresis
	uint8_t* pixel_classification;
	uint8_t* pixel_classification_d;
	pixel_classification = (uint8_t*)malloc(width*height);
	hipMalloc(&pixel_classification_d, width*height*sizeof(uint8_t));

	float_threshold<<<grid, threads>>>(height, width, pixel_classification_d, suppr_mag_d);

	hipMemcpy(pixel_classification, pixel_classification_d, width*height, hipMemcpyDeviceToHost);
	
	stbi_write_png("./output_GPU/4_thresholded.png", width, height, 1, pixel_classification, width);


	hysteresis<<<grid, threads>>>(height, width, pixel_classification_d);

	hipMemcpy(pixel_classification, pixel_classification_d, width*height, hipMemcpyDeviceToHost);

	stbi_write_png("./output_GPU/5_hysteresis.png", width, height, 1, pixel_classification, width);


	//Dilation and Erosion

	//Dilation

	

	uint8_t* dilation;
	uint8_t* dilation_d;
	dilation = (uint8_t*)malloc(width*height);
	hipMalloc(&dilation_d, width*height*sizeof(uint8_t));

	//dilation kernel
	int dilation_kernel_size = 3;
	float dilation_kernel[dilation_kernel_size * dilation_kernel_size] = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f};
	float* dilation_kernel_d;
	hipMalloc(&dilation_kernel_d, dilation_kernel_size * dilation_kernel_size * sizeof(float));
	hipMemcpy(dilation_kernel_d, dilation_kernel, dilation_kernel_size * dilation_kernel_size * sizeof(float), hipMemcpyHostToDevice);


	printf("grids: %d, %d\n", grid.x, grid.y);
	printf("threads: %d, %d\n", threads.x, threads.y);
	apply_dilation<<<grid, threads, 0>>>(dilation_kernel_size, height, width, dilation_d, pixel_classification_d, dilation_kernel_d);


	hipMemcpy(dilation, dilation_d, width*height, hipMemcpyDeviceToHost);

	// for (int i = 0; i < width*height; i++){
	// 		if(dilation[i] != 0)
	// 		printf("i = %d, dilation = %d\n", i, dilation[i]);

	// }

	stbi_write_png("./output_GPU/6_dilation.png", width, height, 1, dilation, width);

	// //Erosion

	uint8_t* erosion;
	uint8_t* erosion_d;
	erosion = (uint8_t*)malloc(width*height);
	hipMalloc(&erosion_d, width*height);

	//erosion kernel
	int erosion_kernel_size = 3;
	float erosion_kernel[erosion_kernel_size*erosion_kernel_size] = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f};
	float* erosion_kernel_d;
	hipMalloc(&erosion_kernel_d, erosion_kernel_size*erosion_kernel_size*sizeof(float));

	hipMemcpy(erosion_kernel_d, erosion_kernel, erosion_kernel_size*erosion_kernel_size*sizeof(float), hipMemcpyHostToDevice);


	apply_erosion<<<grid, threads>>>(erosion_kernel_size, height, width, erosion_d, dilation_d, erosion_kernel_d);

	hipMemcpy(erosion, erosion_d, width*height, hipMemcpyDeviceToHost);

	stbi_write_png("./output_GPU/7_erosion.png", width, height, 1, erosion, width);
    return 0;
}