#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cstdlib>
#include <cmath>
#include <algorithm>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define BLOCK_SIZE 16


//Should be the threshold based on the max magnitudes seen in the image. In our case most likely 255

#define LAPLACIAN_GAUSSIAN 1
#define GAUSSIAN_KERNEL_SIZE 3
#define GAUSSIAN_SIGMA 1.2
#define SHARED 0



#define MAX_THRESHOLD_MULT 0.15
#define MIN_THRESHOLD_MULT 0.02
#define NON_MAX_SUPPR_THRESHOLD 0.4



__global__ void apply_filter_global(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel){
	
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;


	if(i < height && j < width){
		float sum = 0.0f; // Initialize sum to 0 inside the loop

		for (int k = 0; k < kernel_size; k++)
		{
			for (int m = 0; m < kernel_size; m++)
			{
				int input_row = i + (k - 1);
				int input_col = j + (m - 1);

				// Check if the indices are within bounds
				if (input_row >= 0 && input_row < height && input_col >= 0 && input_col < width) {
					sum += kernel[k * kernel_size + m] * input[input_row * width + input_col];
				}
			}
		}

		//printf("%f\n", sum);
		output[i * width + j] = (uint8_t)abs(sum);
	}
}


__global__ void apply_filter_shared(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel){
	extern __shared__ float kernel_shared[];

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (blockIdx.x < kernel_size && blockIdx.y < kernel_size) {
		kernel_shared[threadIdx.y * kernel_size + threadIdx.x] = kernel[threadIdx.y * kernel_size + threadIdx.x];
	}

	__syncthreads(); // Ensure all threads have finished copying to shared memory

	// if(threadIdx.x == 0 && threadIdx.y == 0){
	// 	printf("kernel size: %d\n", kernel_size);
	// 	for(int i = 0; i < kernel_size; i++){
	// 		for(int j = 0; j < kernel_size; j++){
	// 			printf("%f ", kernel[i*kernel_size + j]);
	// 		}
	// 		printf("\n");
	// 	}
	// }

	if (i < height && j < width) {
		float sum = 0;

		// printf("%d, ", input[i*width + j]);
		//printf("%f, %f, %f, %f, %f, %f, %f, %f, %f\n", kernel_shared[0], kernel_shared[1], kernel_shared[2], kernel_shared[3], kernel_shared[4], kernel_shared[5], kernel_shared[6], kernel_shared[7], kernel_shared[8]);
		for (int k = 0; k < kernel_size; k++) {
			for (int m = 0; m < kernel_size; m++) {
				int input_row = i + (k - 1);
				int input_col = j + (m - 1);

				// Check if the indices are within bounds
				if (input_row >= 0 && input_row < height && input_col >= 0 && input_col < width) {
					sum += kernel_shared[k * kernel_size + m] * input[input_row * width + input_col];
					if(threadIdx.x == 1 && threadIdx.y == 1){
						//printf("k = %d, m= %d, kernel_shared -> %f, input -> %d\n",k,m, kernel_shared[k * kernel_size + m], input[input_row * width + input_col]);
					}
				}
			}
		}
		

		output[i * width + j] = (uint8_t)abs(sum);
	}
}


__global__ void apply_filter_shared_tiled(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel){
	
}

__global__ void convert_to_greyscale(int height, int width, uint8_t *img, uint8_t *grey_img)
{

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < height && j < width){
		auto b = img[i*width*3 + j*3 + 0];
		auto g = img[i*width*3 + j*3 + 1];
		auto r = img[i*width*3 + j*3 + 2];

		int average = (int)(0.3*r + 0.59*g + 0.11*b); // Adjust the weights for each channel

		grey_img[i*width + j] = average;
	}
}

__global__ void compute_magnitude_and_gradient(int height, int width, uint8_t *Ix, uint8_t *Iy, uint8_t *mag, float *grad){

	//can we use shared memory for this??

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < height && j < width){
		float dx = Ix[i*width+j];
		float dy = Iy[i*width+j];
		mag[i*width+j] = (int)sqrt(dx*dx+dy*dy);
		float angle = atan2(dy, dx)*180/M_PI;
		grad[i*width+j] = angle < 180 ? angle+180 : angle;
	}
	

}

__global__ void non_maximum_suppression(int height, int width, uint8_t *suppr_mag, uint8_t *mag, float* grad){

	// for(int i = 0; i < height; i++)
 	// {
    //  	for(int j = 0; j < width; j++)
	//  	{
	// 		// in cpp is there a better way to initialize with zeros??
	// 		suppr_mag[i*width + j] = 0;
    //  	}
 	// }


	// for(int i = 1; i < height-1; i++)
 	// {
    //  	for(int j = 1; j < width-1; j++)
	//  	{
	// 		int q = 255;
	// 		int r = 255;

	// 		//angle 0
    //         if (0 <= grad[i*width+j] < 22.5 || 157.5 <= grad[i*width+j] <= 180){
    //             q = mag[i*width + j+1];
    //             r = mag[i*width + j-1];
	// 		}
    //         //angle 45
    //         else if (22.5 <= grad[i*width+j] < 67.5){
    //             q = mag[(i+1)*width + j-1];
    //             r = mag[(i-1)*width + j+1];
	// 		}
    //         //angle 90
    //         else if (67.5 <= grad[i*width+j] < 112.5){
    //             q = mag[(i+1)*width + j];
    //             r = mag[(i-1)*width + j];
	// 		}
    //         //angle 135
    //         else if (112.5 <= grad[i*width+j] < 157.5){
    //             q = mag[(i-1)*width + j-1];
    //             r = mag[(i+1)*width + j+1];
	// 		}

	// 		if (mag[i*width + j] >= q && mag[i*width + j] >= r){
    //             suppr_mag[i*width + j] = mag[i*width + j];
	// 		} else {
	// 			suppr_mag[i*width + j] = 0;
	// 		}

    //  	}
 	// }



}

__global__ void double_threshold(int height, int width,  uint8_t *pixel_classification,  uint8_t *suppr_mag){

	// float high_threshold = 0.09*255;
	// float low_threshold = high_threshold*0.05;

	// std::cout<<low_threshold<<", "<<high_threshold<<std::endl;
	
	// for(int i = 0; i < height; i++)
 	// {
    //  	for(int j = 0; j < width; j++)
	//  	{
	// 		if(suppr_mag[i*width+j] >= high_threshold){
	// 			// strong pixels
	// 			pixel_classification[i*width+j] = 255;
	// 		} else if (suppr_mag[i*width+j] < low_threshold){
	// 			// non relevant pixels
	// 			pixel_classification[i*width+j] = 0;
	// 		} else {
	// 			// weak pixels
	// 			pixel_classification[i*width+j] = 25;
	// 		}
    //  	}
 	// }



}

__global__ void hysteresis(int height, int width, uint8_t *pixel_classification){

	// for(int i = 1; i < height-1; i++)
 	// {
    //  	for(int j = 1; j < width-1; j++)
	//  	{
	// 		if(pixel_classification[i*width+j] == 25){
	// 			if(pixel_classification[(i+1)*width+j-1] == 255 || pixel_classification[(i+1)*width+j] == 255 || pixel_classification[(i+1)*width+j+1] == 255 ||
	// 			pixel_classification[i*width+j-1] == 255 || pixel_classification[i*width+j+1] == 255 || pixel_classification[(i-1)*width+j-1] == 255 ||
	// 			pixel_classification[(i-1)*width+j] == 255 || pixel_classification[(i-1)*width+j+1] == 255){
	// 				pixel_classification[i*width + j] = 255;
	// 			} else {
	// 				pixel_classification[i*width + j] = 0;
	// 			}
	// 		}
    //  	}
 	// }
    


}


// void measure_time(bool start, FILE* file_times, std::string name){
// 	static std::chrono::system_clock::time_point start_time;
// 	static std::chrono::system_clock::time_point end_time;
// 	if(start){
// 		start_time = std::chrono::system_clock::now();
// 	} else {
// 		end_time = std::chrono::system_clock::now();
// 		std::chrono::duration<double> duration = end_time - start_time;
// 		fprintf(file_times, "%s: %f \n", name.c_str(), duration.count());
// 	}
// }

float* get_gaussian_filter (int kernel_size, float sigma){

	kernel_size = kernel_size%2 == 0 ? kernel_size-1 : kernel_size;

	float* gaussian_filter = (float*)malloc(kernel_size*kernel_size*sizeof(float));
	float sum = 0.0;
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
			sum += gaussian_filter[i*kernel_size + j];
		}
	}
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] /= sum;
		}
	}
	return gaussian_filter;

}

float* get_gaussian_laplacian_filter (int kernel_size, float sigma){
	float* gaussian_filter = (float*)malloc(kernel_size*kernel_size*sizeof(float));
	float sum = 0.0;
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] = (((i*i+j*j)/(2*sigma*sigma))-1)*exp(-(i*i+j*j)/(2*sigma*sigma))/(M_PI*sigma*sigma*sigma*sigma);
			sum += gaussian_filter[i*kernel_size + j];
		}
	}
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] /= sum;
		}
	}
	return gaussian_filter;


}



int main(int argc, char *argv[])
{
    //Cuda definitions
    const int blocksize = BLOCK_SIZE;
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    int device;
    struct hipDeviceProp_t properties;
    

    hipError_t err = hipSuccess;
    hipDeviceProp_t deviceProp;
    int devID = 0;
    auto error = hipGetDevice(&devID);

    if (error != hipSuccess) {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited) {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_FAILURE);
    }

    if (error != hipSuccess) {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    dim3 threads, grid;

    
    //image definitions
    int width, height, bpp;
    
	auto img_fname = argc>=2 ? argv[1] : "image.png";

	system("mkdir -p output_GPU");
	auto file_times = fopen("./output/times.txt", "w");
	

    //program starts

	uint8_t* rgb_image = stbi_load(img_fname, &width, &height, &bpp, 3);
    uint8_t* rgb_image_d;
    hipMalloc(&rgb_image_d, width*height*3);
    hipMemcpy(rgb_image_d, rgb_image, width*height*3, hipMemcpyHostToDevice);

	threads = dim3(blocksize, blocksize);
    grid = dim3((width + threads.x - 1) / threads.x , (height + threads.y - 1) / threads.y);
    printf("CUDA kernel launch with %d blocks of %d threads\n", grid.x * grid.y, threads.x * threads.y);



    std::cout<<"image: "<<img_fname<<std::endl;
	std::cout<<width<<" "<<height<<std::endl;

    //Stop here

	// Convert to greyscale
	uint8_t* grey_image;
	uint8_t* grey_image_d;
	grey_image = (uint8_t*)malloc(width*height);
	hipMalloc(&grey_image_d, width*height);

	//measure_time(true, file_times, "convert_to_greyscale");
	// convert_to_greyscale(height, width, rgb_image, grey_image);
	convert_to_greyscale<<<grid, threads>>>(height, width, rgb_image_d, grey_image_d);
	hipMemcpy(grey_image, grey_image_d, width*height, hipMemcpyDeviceToHost);

	stbi_image_free(rgb_image);
	hipFree(rgb_image_d);
	stbi_write_png("./output_GPU/0_image_grey.png", width, height, 1, grey_image, width);

	
	// Apply Gaussian filtering
	
	auto kernel_size = GAUSSIAN_KERNEL_SIZE;
	float sigma = GAUSSIAN_SIGMA;
	#if LAPLACIAN_GAUSSIAN
		float* gaussian_filter = get_gaussian_laplacian_filter(kernel_size, sigma);
	#else
		float* gaussian_filter = get_gaussian_filter(kernel_size, sigma);
	#endif
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			std::cout<<gaussian_filter[i*kernel_size + j]<<" ";
		}
		std::cout<<std::endl;
	}

	uint8_t* gaussian_image;
	uint8_t* gaussian_image_d;
	float* gaussian_filter_d;
    gaussian_image = (uint8_t*)malloc(width*height);
	hipMalloc(&gaussian_image_d, width*height);
	hipMalloc(&gaussian_filter_d, kernel_size*kernel_size*sizeof(float));
	hipMemcpy(gaussian_filter_d, gaussian_filter, kernel_size*kernel_size*sizeof(float), hipMemcpyHostToDevice);	

	#if SHARED
		apply_filter_shared<<<grid, threads, kernel_size*kernel_size*sizeof(float)>>>(kernel_size, height, width, gaussian_image_d, grey_image_d, gaussian_filter_d);
	#else
		apply_filter_global<<<grid, threads>>>(kernel_size, height, width, gaussian_image_d, grey_image_d, gaussian_filter_d);
	#endif

	hipMemcpy(gaussian_image, gaussian_image_d, width*height, hipMemcpyDeviceToHost);
	stbi_image_free(grey_image);
	stbi_write_png("./output_GPU/0_image_gaussian.png", width, height, 1, gaussian_image, width);

	

	//Apply 3x3 Sobel filtering
	float sobel_h[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
	float sobel_v[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
	uint8_t* sobel_image_h;
	uint8_t* sobel_image_v;
	uint8_t* sobel_image_h_d;
	uint8_t* sobel_image_v_d;
	float* sobel_h_d;
	float* sobel_v_d;

	sobel_image_h = (uint8_t*)malloc(width*height);
	sobel_image_v = (uint8_t*)malloc(width*height);
	hipMalloc(&sobel_image_h_d, width*height);
	hipMalloc(&sobel_image_v_d, width*height);
	hipMalloc(&sobel_h_d, 9*sizeof(float));
	hipMalloc(&sobel_v_d, 9*sizeof(float));

	hipMemcpy(sobel_h_d, sobel_h, 9*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(sobel_v_d, sobel_v, 9*sizeof(float), hipMemcpyHostToDevice);

	#if SHARED
		apply_filter_shared<<<grid, threads, 9*sizeof(float)>>>(3, height, width, sobel_image_h_d, gaussian_image_d, sobel_h_d);
		apply_filter_shared<<<grid, threads, 9*sizeof(float)>>>(3, height, width, sobel_image_v_d, gaussian_image_d, sobel_v_d);
	#else
		apply_filter_global<<<grid, threads>>>(3, height, width, sobel_image_h_d, gaussian_image_d, sobel_h_d);
		apply_filter_global<<<grid, threads>>>(3, height, width, sobel_image_v_d, gaussian_image_d, sobel_v_d);
	#endif

	hipMemcpy(sobel_image_h, sobel_image_h_d, width*height, hipMemcpyDeviceToHost);
	hipMemcpy(sobel_image_v, sobel_image_v_d, width*height, hipMemcpyDeviceToHost);

	stbi_image_free(gaussian_image);
	stbi_write_png("./output_GPU/1_image_sobel_h.png", width, height, 1, sobel_image_h, width);
	stbi_write_png("./output_GPU/1_image_sobel_v.png", width, height, 1, sobel_image_v, width);




	// // Calculate magnitude and gradient direction
    // float* gradient_direction;
    // gradient_direction = (float*)malloc(width*height*sizeof(float));
	// uint8_t* magnitude;
    // magnitude = (uint8_t*)malloc(width*height);

	// measure_time(true, file_times, "compute_magnitude_and_gradient");
	// compute_magnitude_and_gradient(height, width, sobel_image_h, sobel_image_v, magnitude, gradient_direction);
	// measure_time(false, file_times, "compute_magnitude_and_gradient");

	// stbi_image_free(sobel_image_v);
	// stbi_image_free(sobel_image_h);
	// stbi_write_png("./output/2_gradient_direction.png", width, height, 1, gradient_direction, width);
	// stbi_write_png("./output/2_magnitude.png", width, height, 1, magnitude, width);

	float* gradient_direction;
	float* gradient_direction_d;
	uint8_t* magnitude;
	uint8_t* magnitude_d;
	
	gradient_direction = (float*)malloc(width*height*sizeof(float));
	magnitude = (uint8_t*)malloc(width*height);
	hipMalloc(&gradient_direction_d, width*height*sizeof(float));
	hipMalloc(&magnitude_d, width*height);

	compute_magnitude_and_gradient<<<grid, threads>>>(height, width, sobel_image_h_d, sobel_image_v_d, magnitude_d, gradient_direction_d);

	hipMemcpy(gradient_direction, gradient_direction_d, width*height*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(magnitude, magnitude_d, width*height, hipMemcpyDeviceToHost);

	stbi_image_free(sobel_image_v);
	stbi_image_free(sobel_image_h);
	stbi_write_png("./output_GPU/2_gradient_direction.png", width, height, 1, gradient_direction, width);
	stbi_write_png("./output_GPU/2_magnitude.png", width, height, 1, magnitude, width);

	// // Non-maximum suppression
	// uint8_t* suppr_mag;
    // suppr_mag = (uint8_t*)malloc(width*height);

	// measure_time(true, file_times, "non_maximum_suppression");
	// non_maximum_suppression(height, width, suppr_mag, magnitude, gradient_direction);
	// measure_time(false, file_times, "non_maximum_suppression");

	// double max = *std::max_element(magnitude, magnitude + width*height);

	// std::cout<<max<<std::endl;

	// stbi_image_free(magnitude);
	// stbi_image_free(gradient_direction);
	// stbi_write_png("./output/3_nonmax_suppr.png", width, height, 1, suppr_mag, width);

	// // classify pixels as strong, weak or non-relevant
	// uint8_t* pixel_classification;
    // pixel_classification = (uint8_t*)malloc(width*height);
	
	// measure_time(true, file_times, "double_threshold");
	// double_threshold(height, width, pixel_classification, suppr_mag);
	// measure_time(false, file_times, "double_threshold");

	// stbi_write_png("./output/4_thresholded.png", width, height, 1, pixel_classification, width);

	// measure_time(true, file_times, "hysteresis");
	// hysteresis(height, width, pixel_classification);
	// measure_time(false, file_times, "hysteresis");

	// stbi_write_png("./output/5_hysteresis.png", width, height, 1, pixel_classification, width);

    return 0;
}


// TODOS:
// zeit messen