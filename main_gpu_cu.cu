#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cstdlib>
#include <cmath>
#include <algorithm>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define BLOCK_SIZE 16


//Should be the threshold based on the max magnitudes seen in the image. In our case most likely 255

#define LAPLACIAN_GAUSSIAN 0
#define GAUSSIAN_KERNEL_SIZE 3
#define GAUSSIAN_SIGMA 1.1




#define MAX_THRESHOLD_MULT 0.15
#define MIN_THRESHOLD_MULT 0.02
#define NON_MAX_SUPPR_THRESHOLD 0.4



__global__ void apply_filter_global(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel){
	
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	// Print the matrix kernel
	//  if (i == 1 && j == 1 ){
	//  	printf("Kernel: \n");
	// 	for (int p = 0; p < kernel_size; p++) {
	// 		for (int o = 0; o < kernel_size; o++) {
	// 			printf("%f ", kernel[p * kernel_size + o]);
	// 		}
	// 	}
	// 	printf("\n");
	//  }

	//print input image 
	//printf("%u ", input[i * width + j]);

	float sum = 0;

	
	if(i < height && j < width){
		for (int k = 0; k < kernel_size; k++)
		{
			for (int m = 0; m < kernel_size; m++)
			{
				sum += kernel[k * kernel_size + m] * input[(i + (k - 1)) * width + j + (m - 1)];
				//sum +=  input[i * width + j];
			}
		}
	}

	//printf("sum: %f\n", sum);
	output[i * width + j] = (int)abs(sum);
	if(output[i * width + j] != 0)
		printf("output: %u\n", output[i * width + j]);
}

__global__ void apply_filter_shared(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel){
	extern __shared__ float kernel_shared[];

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	// for(int p = 0; p < kernel_size; p++){
	// 	for(int o = 0; o < kernel_size; o++){
	// 		printf("%f ", kernel[p*kernel_size + o]);
	// 	}
	// 	printf("\n");
	// }


	if(blockIdx.x < kernel_size && blockIdx.y < kernel_size){
		kernel_shared[threadIdx.y*kernel_size + threadIdx.x] = kernel[threadIdx.y*kernel_size + threadIdx.x];
	}

	if(i == 0 && j==0)
	for(int p = 0; p < kernel_size; p++){
		for(int o = 0; o < kernel_size; o++){
			printf("%f ", kernel_shared[p*kernel_size + o]);
		}
		printf("\n");
	}
	

	__syncthreads();


	if(i < height && j < width){
		float sum = 0;

		for (int k = 0; k < kernel_size; k++)
		{
			for (int m = 0; m < kernel_size; m++)
			{
				sum += kernel_shared[k * kernel_size + m] * input[(i + (k - 1)) * width + j + (m - 1)];
				//sum +=  input[i * width + j];
			}
		}
		output[i * width + j] = (int)abs(sum);
	}

}

__global__ void apply_filter_shared_tiled(int kernel_size, int height, int width, uint8_t *output, uint8_t *input, float *kernel){
	
}

__global__ void convert_to_greyscale(int height, int width, uint8_t *img, uint8_t *grey_img)
{

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < height && j < width){
		auto b = img[i*width*3 + j*3 + 0];
		auto g = img[i*width*3 + j*3 + 1];
		auto r = img[i*width*3 + j*3 + 2];

		int average = (int)(0.3*r + 0.59*g + 0.11*b); // Adjust the weights for each channel

		grey_img[i*width + j] = average;
	}
}

__global__ void compute_magnitude_and_gradient(int height, int width, uint8_t *Ix, uint8_t *Iy, uint8_t *mag, float *grad){

	// for(int i = 1; i < height-1; i++)
 	// {
    //  	for(int j = 1; j < width-1; j++)
	//  	{
	// 		float dx = Ix[i*width+j];
	// 		float dy = Iy[i*width+j];
	// 		mag[i*width+j] = (int)sqrt(dx*dx+dy*dy);
	// 		float angle = atan2(dy, dx)*180/M_PI;
	// 		grad[i*width+j] = angle < 180 ? angle+180 : angle;
    //  	}
 	// }

	

}

__global__ void non_maximum_suppression(int height, int width, uint8_t *suppr_mag, uint8_t *mag, float* grad){

	// for(int i = 0; i < height; i++)
 	// {
    //  	for(int j = 0; j < width; j++)
	//  	{
	// 		// in cpp is there a better way to initialize with zeros??
	// 		suppr_mag[i*width + j] = 0;
    //  	}
 	// }


	// for(int i = 1; i < height-1; i++)
 	// {
    //  	for(int j = 1; j < width-1; j++)
	//  	{
	// 		int q = 255;
	// 		int r = 255;

	// 		//angle 0
    //         if (0 <= grad[i*width+j] < 22.5 || 157.5 <= grad[i*width+j] <= 180){
    //             q = mag[i*width + j+1];
    //             r = mag[i*width + j-1];
	// 		}
    //         //angle 45
    //         else if (22.5 <= grad[i*width+j] < 67.5){
    //             q = mag[(i+1)*width + j-1];
    //             r = mag[(i-1)*width + j+1];
	// 		}
    //         //angle 90
    //         else if (67.5 <= grad[i*width+j] < 112.5){
    //             q = mag[(i+1)*width + j];
    //             r = mag[(i-1)*width + j];
	// 		}
    //         //angle 135
    //         else if (112.5 <= grad[i*width+j] < 157.5){
    //             q = mag[(i-1)*width + j-1];
    //             r = mag[(i+1)*width + j+1];
	// 		}

	// 		if (mag[i*width + j] >= q && mag[i*width + j] >= r){
    //             suppr_mag[i*width + j] = mag[i*width + j];
	// 		} else {
	// 			suppr_mag[i*width + j] = 0;
	// 		}

    //  	}
 	// }



}

__global__ void double_threshold(int height, int width,  uint8_t *pixel_classification,  uint8_t *suppr_mag){

	// float high_threshold = 0.09*255;
	// float low_threshold = high_threshold*0.05;

	// std::cout<<low_threshold<<", "<<high_threshold<<std::endl;
	
	// for(int i = 0; i < height; i++)
 	// {
    //  	for(int j = 0; j < width; j++)
	//  	{
	// 		if(suppr_mag[i*width+j] >= high_threshold){
	// 			// strong pixels
	// 			pixel_classification[i*width+j] = 255;
	// 		} else if (suppr_mag[i*width+j] < low_threshold){
	// 			// non relevant pixels
	// 			pixel_classification[i*width+j] = 0;
	// 		} else {
	// 			// weak pixels
	// 			pixel_classification[i*width+j] = 25;
	// 		}
    //  	}
 	// }



}

__global__ void hysteresis(int height, int width, uint8_t *pixel_classification){

	// for(int i = 1; i < height-1; i++)
 	// {
    //  	for(int j = 1; j < width-1; j++)
	//  	{
	// 		if(pixel_classification[i*width+j] == 25){
	// 			if(pixel_classification[(i+1)*width+j-1] == 255 || pixel_classification[(i+1)*width+j] == 255 || pixel_classification[(i+1)*width+j+1] == 255 ||
	// 			pixel_classification[i*width+j-1] == 255 || pixel_classification[i*width+j+1] == 255 || pixel_classification[(i-1)*width+j-1] == 255 ||
	// 			pixel_classification[(i-1)*width+j] == 255 || pixel_classification[(i-1)*width+j+1] == 255){
	// 				pixel_classification[i*width + j] = 255;
	// 			} else {
	// 				pixel_classification[i*width + j] = 0;
	// 			}
	// 		}
    //  	}
 	// }
    


}


// void measure_time(bool start, FILE* file_times, std::string name){
// 	static std::chrono::system_clock::time_point start_time;
// 	static std::chrono::system_clock::time_point end_time;
// 	if(start){
// 		start_time = std::chrono::system_clock::now();
// 	} else {
// 		end_time = std::chrono::system_clock::now();
// 		std::chrono::duration<double> duration = end_time - start_time;
// 		fprintf(file_times, "%s: %f \n", name.c_str(), duration.count());
// 	}
// }

float* get_gaussian_filter (int kernel_size, float sigma){

	kernel_size = kernel_size%2 == 0 ? kernel_size-1 : kernel_size;

	float* gaussian_filter = (float*)malloc(kernel_size*kernel_size*sizeof(float));
	float sum = 0.0;
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
			sum += gaussian_filter[i*kernel_size + j];
		}
	}
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] /= sum;
		}
	}
	return gaussian_filter;

}

float* get_gaussian_laplacian_filter (int kernel_size, float sigma){
	float* gaussian_filter = (float*)malloc(kernel_size*kernel_size*sizeof(float));
	float sum = 0.0;
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] = (((i*i+j*j)/(2*sigma*sigma))-1)*exp(-(i*i+j*j)/(2*sigma*sigma))/(M_PI*sigma*sigma*sigma*sigma);
			sum += gaussian_filter[i*kernel_size + j];
		}
	}
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			gaussian_filter[i*kernel_size + j] /= sum;
		}
	}
	return gaussian_filter;


}



int main(int argc, char *argv[])
{
    //Cuda definitions
    const int blocksize = BLOCK_SIZE;
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    int device;
    struct hipDeviceProp_t properties;
    

    hipError_t err = hipSuccess;
    hipDeviceProp_t deviceProp;
    int devID = 0;
    auto error = hipGetDevice(&devID);

    if (error != hipSuccess) {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited) {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_FAILURE);
    }

    if (error != hipSuccess) {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    dim3 threads, grid;

    
    //image definitions
    int width, height, bpp;
    
	auto img_fname = argc>=2 ? argv[1] : "image.png";

	system("mkdir -p output_GPU");
	auto file_times = fopen("./output/times.txt", "w");
	

    //program starts

	uint8_t* rgb_image = stbi_load(img_fname, &width, &height, &bpp, 3);
    uint8_t* rgb_image_d;
    hipMalloc(&rgb_image_d, width*height*3);
    hipMemcpy(rgb_image_d, rgb_image, width*height*3, hipMemcpyHostToDevice);

	threads = dim3(blocksize, blocksize);
    grid = dim3((width + threads.x - 1) / threads.x , (height + threads.y - 1) / threads.y);
    printf("CUDA kernel launch with %d blocks of %d threads\n", grid.x * grid.y, threads.x * threads.y);



    std::cout<<"image: "<<img_fname<<std::endl;
	std::cout<<width<<" "<<height<<std::endl;

    //Stop here

	// Convert to greyscale
	uint8_t* grey_image;
	uint8_t* grey_image_d;
	grey_image = (uint8_t*)malloc(width*height);
	hipMalloc(&grey_image_d, width*height);

	//measure_time(true, file_times, "convert_to_greyscale");
	// convert_to_greyscale(height, width, rgb_image, grey_image);
	convert_to_greyscale<<<grid, threads>>>(height, width, rgb_image_d, grey_image_d);
	hipDeviceSynchronize(); // Synchronize with CUDA
	hipMemcpy(grey_image, grey_image_d, width*height, hipMemcpyDeviceToHost);

	//print
	// for(int i = 0; i < height; i++){
	// 	for(int j = 0; j < width; j++){
	// 		printf("%d ", grey_image[i*width + j]);
	// 	}
	// 	printf("\n");
	// }

	//measure_time(false, file_times, "convert_to_greyscale");
	stbi_image_free(rgb_image);
	hipFree(rgb_image_d);
	stbi_write_png("./output_GPU/0_image_grey.png", width, height, 1, grey_image, width);

	


	// Apply Gaussian filtering
	
	auto kernel_size = GAUSSIAN_KERNEL_SIZE;
	float sigma = GAUSSIAN_SIGMA;
	#if LAPLACIAN_GAUSSIAN
		float* gaussian_filter = get_gaussian_laplacian_filter(kernel_size, sigma);
	#else
		float* gaussian_filter = get_gaussian_filter(kernel_size, sigma);
	#endif
	//float gaussian_filter[9] = {1,1,1,1,1,1,1,1,1};
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			std::cout<<gaussian_filter[i*kernel_size + j]<<" ";
		}
		std::cout<<std::endl;
	}

	uint8_t* gaussian_image;
	uint8_t* gaussian_image_d;
	float* gaussian_filter_d;
    gaussian_image = (uint8_t*)malloc(width*height);
	hipMalloc(&gaussian_image_d, width*height);
	hipMalloc(&gaussian_filter_d, kernel_size*kernel_size*sizeof(float));
	printf("Copying input data from the host memory to the CUDA device\n");
	hipMemcpy(gaussian_filter_d, gaussian_filter, kernel_size*kernel_size*sizeof(float), hipMemcpyHostToDevice);	
	printf("copied");
	// static global memory version
	apply_filter_global<<<grid, threads>>>(kernel_size, height, width, gaussian_image_d, grey_image_d, gaussian_filter_d);

	//dinamic shared memory version
	// apply_filter_shared<<<grid, threads, kernel_size*kernel_size*sizeof(float)>>>(kernel_size, height, width, gaussian_image_d, grey_image_d, gaussian_filter_d);


	hipMemcpy(gaussian_image, gaussian_image_d, width*height, hipMemcpyDeviceToHost);
	//print image 
	// for(int i = 0; i < height; i++){
	// 	for(int j = 0; j < width; j++){
	// 		printf("%u ", gaussian_image[i*width + j]);
	// 	}
	// 	printf("\n");
	// }

	//free(gaussian_filter);
	stbi_image_free(grey_image);
	stbi_write_png("./output_GPU/0_image_gaussian.png", width, height, 1, gaussian_image, width);


	// measure_time(true, file_times, "apply_gaussian_filter");
	// apply_filter(kernel_size, height, width, gaussian_image, grey_image, gaussian_filter);
	// measure_time(false, file_times, "apply_gaussian_filter");

	// stbi_image_free(grey_image);
	// stbi_write_png("./output/0_image_gaussian.png", width, height, 1, gaussian_image, width);
	
	// // Apply Sobel filtering
	// float sobel_h[kernel_size*kernel_size] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
	// float sobel_v[kernel_size*kernel_size] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
	// uint8_t* sobel_image_h;
    // sobel_image_h = (uint8_t*)malloc(width*height);
	// uint8_t* sobel_image_v;
    // sobel_image_v = (uint8_t*)malloc(width*height);

	// measure_time(true, file_times, "apply_sobel_filters");
	// apply_filter(kernel_size, height, width, sobel_image_h, gaussian_image, sobel_h);
	// apply_filter(kernel_size, height, width, sobel_image_v, gaussian_image, sobel_v);
	// measure_time(false, file_times, "apply_sobel_filters");

    // stbi_image_free(gaussian_image);
	// stbi_write_png("./output/1_image_sobel_h.png", width, height, 1, sobel_image_h, width);
	// stbi_write_png("./output/1_image_sobel_v.png", width, height, 1, sobel_image_v, width);

	// // Calculate magnitude and gradient direction
    // float* gradient_direction;
    // gradient_direction = (float*)malloc(width*height*sizeof(float));
	// uint8_t* magnitude;
    // magnitude = (uint8_t*)malloc(width*height);

	// measure_time(true, file_times, "compute_magnitude_and_gradient");
	// compute_magnitude_and_gradient(height, width, sobel_image_h, sobel_image_v, magnitude, gradient_direction);
	// measure_time(false, file_times, "compute_magnitude_and_gradient");

	// stbi_image_free(sobel_image_v);
	// stbi_image_free(sobel_image_h);
	// stbi_write_png("./output/2_gradient_direction.png", width, height, 1, gradient_direction, width);
	// stbi_write_png("./output/2_magnitude.png", width, height, 1, magnitude, width);

	// // Non-maximum suppression
	// uint8_t* suppr_mag;
    // suppr_mag = (uint8_t*)malloc(width*height);

	// measure_time(true, file_times, "non_maximum_suppression");
	// non_maximum_suppression(height, width, suppr_mag, magnitude, gradient_direction);
	// measure_time(false, file_times, "non_maximum_suppression");

	// double max = *std::max_element(magnitude, magnitude + width*height);

	// std::cout<<max<<std::endl;

	// stbi_image_free(magnitude);
	// stbi_image_free(gradient_direction);
	// stbi_write_png("./output/3_nonmax_suppr.png", width, height, 1, suppr_mag, width);

	// // classify pixels as strong, weak or non-relevant
	// uint8_t* pixel_classification;
    // pixel_classification = (uint8_t*)malloc(width*height);
	
	// measure_time(true, file_times, "double_threshold");
	// double_threshold(height, width, pixel_classification, suppr_mag);
	// measure_time(false, file_times, "double_threshold");

	// stbi_write_png("./output/4_thresholded.png", width, height, 1, pixel_classification, width);

	// measure_time(true, file_times, "hysteresis");
	// hysteresis(height, width, pixel_classification);
	// measure_time(false, file_times, "hysteresis");

	// stbi_write_png("./output/5_hysteresis.png", width, height, 1, pixel_classification, width);

    return 0;
}


// TODOS:
// zeit messen